#include "hip/hip_runtime.h"
//Name Removed

// Homework 1
// Color to Greyscale Conversion

#include "reference_calc.cpp"
#include "utils.h"
#include <stdio.h>

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
  
  //get thread/block indexes
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy = blockIdx.y * blockDim.y + threadIdx.y;
  
  //get pixel index
  int pix_index = (idx * numCols) + idy;
  
  //grab rgb data
  const uchar4 input_val = rgbaImage[pix_index];
  
  //greyscale the output
  greyImage[pix_index] = input_val.x * .299f + input_val.y * .587f + input_val.z * .114f;
}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
  
  // using 256 (16 * 16) thread per block
  // blocks should be picture divided into blocks of 256 pixels (rounded up), divided by 16 in x & y directions
  
  const dim3 gridSize(16,16,1);
  const dim3 blockSize(((numRows+15)/16),((numCols+15)/16),1);
  rgba_to_greyscale<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
  
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}
