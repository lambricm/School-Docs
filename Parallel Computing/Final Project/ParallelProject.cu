#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <vector>
#include <algorithm>
#include <numeric>
#include <typeinfo>
#include "./parallel_code.cu"

template <class T>
T sum(std::vector<T> data) {
	T result = std::accumulate(data.begin(), data.end(), T());
	return result;
}

float average(std::vector<std::string> data) {
	std::cout << "Invalid: Cannot average string type" << std::endl;
	return 0.0;
}

float average(std::vector<int> data) {
	float result = 1.0 * sum(data) / data.size();
	return result;
}

float average(std::vector<float> data) {
	float result = 1.0 * sum(data) / data.size();
	return result;
}

template <class T>
T min(std::vector<T> data) {
	T result = *(std::min_element(data.begin(), data.end()));
	return result;
}

template <class T>
T max(std::vector<T> data) {
	T result = *(std::max_element(data.begin(), data.end()));
	return result;
}

std::vector<std::string> scan(std::vector<std::string> data, bool inclusive)
{
	std::cout << "Invalid: Cannot scan strings" << std::endl;
	return data;
}

std::vector<float> scan(std::vector<float> data, bool inclusive)
{
	std::vector<float> results;
	if (inclusive)
	{
		for (int i = 0; i < data.size(); i++)
		{
			if (i > 0)
			{
				results.push_back(data[i] + results[i - 1]);
			}
			else
			{
				results.push_back(data[i]);
			}
		}
	}
	else
	{
		std::vector<float> inclusiveResults = scan(data, true);
		for (int i = 0; i < data.size(); i++)
		{
			results.push_back(inclusiveResults[i] - data[i]);
		}
	}
	return results;
}

std::vector<int> scan(std::vector<int> data, bool inclusive)
{
	std::vector<int> results;
	if (inclusive)
	{
		for (int i = 0; i < data.size(); i++)
		{
			if (i > 0)
			{
				results.push_back(data[i] + results[i - 1]);
			}
			else
			{
				results.push_back(data[i]);
			}
		}
	}
	else
	{
		std::vector<int> inclusiveResults = scan(data, true);
		for (int i = 0; i < data.size(); i++)
		{
			results.push_back(inclusiveResults[i] - data[i]);
		}
	}
	return results;
}

template <class T>
std::vector<T> split(const std::string& s, char c, std::vector<T> v, int colOfInterest = -1) {
	int i = 0;
	int j = s.find(c);
	std::vector<T> splitData;

	while (j >= 0) {
		//if (T == int)
		//{
		std::stringstream is(s.substr(i, j-i));
		//}
		T tempVal;
		is >> tempVal;
		splitData.push_back(tempVal);
		i = ++j;
		j = s.find(c, j);

		if (j < 0) {
			//if (T == "int")
			//{
			std::stringstream is(s.substr(i, s.length()));
			//}
			T tempVal2;
			is >> tempVal2;
  			splitData.push_back(tempVal2);
		}
	}
	std::vector<T> concatData = v;
	if (colOfInterest == -1)
	{
		concatData.insert(concatData.end(), splitData.begin(), splitData.end());
	}
	else
	{
		concatData.push_back(splitData[colOfInterest]);
	}
	return concatData;
}

template <class T>
std::vector<T> loadCSV(std::istream& in, std::vector<T> data, int colOfInterest, bool headersOnly = false) {

	std::vector<T> loadedData = data;

	std::string tmp;

	if (headersOnly)
	{
		getline(in, tmp, '\n');
		
		loadedData = split<T>(tmp, ',', loadedData, colOfInterest);
	}
	else
	{
		getline(in, tmp, '\n');
		getline(in, tmp, '\n');
		tmp.clear();
		while (!in.eof()) {
			getline(in, tmp, '\n');

			loadedData = split<T>(tmp, ',', loadedData, colOfInterest);

			tmp.clear();
	
		}
	}
	return loadedData;
}

template <class T>
void printVector(std::vector<T> data)
{
	for (int i = 0; i < data.size(); i++)
	{
		std::cout << i << "\t:" << data[i] << std::endl;
	}
}

void printhelp ()
{
	std::cout << "Syntax: data_filepath function print column_index [serial] [count_value] [hard top threshold] [hard bottom threshold] [soft threshold]" << std::endl;
	std::cout << "function: i (ingest only), s (sum), a (average), m (minimum), M (maximum), c (count of value), n (scan - exclusive), N (scan - inclusive), t (TEEN), d (DSSS)" << std::endl;
	std::cout << "print: r (read only), p (print function result only), P (print data & result of function) s (serial)" << std::endl;
	std::cout << "column_index: 0-based" << std::endl;
	std::cout << "count_value (if applicable)" << std::endl;
	std::cout << "Help Syntax: h" << std::endl;
}

template <class T>
int mainfunc(int argc, char** argv) {
	// Should use an actual arg parse library or something

	// Only runs if filepath is given - maybe should check if filepath is valid
	bool badsyntax = false;
	std::ifstream in(argv[1]);
	T sumValue = T();
	float averageValue = 0.0;
	T minValue = T();
	T maxValue = T();
	int countValue = 0;
	std::vector<T> scanValues;

	bool serial = false;

	if (argc >= 6){
		bool serial = (*argv[5] == 's');
	}

	if (!in)
		return(EXIT_FAILURE);

	std::vector<T> data;
	std::vector<T>* data_ptr = &data;

	data = loadCSV<T>(in, data, atoi(argv[4]));
	
	if (*argv[2] == 'i')
	{
		// do nothing else - just ingest data
	}
	// Sum of data
	else if (*argv[2] == 's')
	{
		if (serial){
			if (typeid(T) != typeid(std::string))
			{
				sumValue = sum(data);
			}
			else
			{
				std::cout << "Invalid: Cannot sum string type" << std::endl;
			}
		}
		else{
			sumValue = parallel::reduce(data_ptr);
		}
	}
	// Average of data
	else if (*argv[2] == 'a')
	{
		if (serial){
			if (typeid(T) != typeid(std::string))
			{
				averageValue = average(data);				
			}
		}
		else{
			averageValue = parallel::average(&data);
		}
	}
	// Min of data
	else if (*argv[2] == 'm')
	{
		if (serial){
			minValue = min(data);
		}
		else{
			minValue = parallel::minimum(&data);
		}
	}
	// Max of data
	else if (*argv[2] == 'M')
	{	if (serial){
			maxValue = max(data);
		}
		else{
			maxValue = parallel::maximum(&data);
		}
	}
	// Count of data
	else if (*argv[2] == 'c' &&  argv[5] != "")
	{

		if(serial){
			std::stringstream ss(argv[6]);
			T countObject;
			ss >> countObject;

			countValue = std::count(data.begin(), data.end(), countObject);
		}
		else{
			std::stringstream ss(argv[5]);
			T countObject;
			ss >> countObject;

			countValue = parallel::count(&data, countObject);
		}
	}
	else if (*argv[2] == 'n')
	{
		if(serial){
			scanValues = scan(data, false);
		}
		else{
			parallel::exclusive_scan(data);
			scanValues = data;
		}
	}
	else if (*argv[2] == 'N')
	{
		if(serial){
			scanValues = scan(data, true);
		}
		else{
			parallel::inclusive_scan(data);
			scanValues = data;
		}
	}
	else if (*argv[2] == 't')
	{
		if (serial){
			std::stringstream ss1(argv[6]);
			T h_top_thres;
			ss1 >> h_top_thres;
			std::stringstream ss2(argv[7]);
			T h_bot_thres;
			ss2 >> h_bot_thres;
			std::stringstream ss3(argv[8]);
			T s_thres;
			ss3 >> s_thres;
			
			std::vector<T>* temp = TEEN(data, h_top_thres, h_bot_thres, s_thres);
			data.clear();
			for (int i = 0; i<temp->size();i++){
				data.push_back(temp->at(i));
			}
		}
		else{
			std::stringstream ss1(argv[5]);
			T h_top_thres;
			ss1 >> h_top_thres;
			std::stringstream ss2(argv[6]);
			T h_bot_thres;
			ss2 >> h_bot_thres;
			std::stringstream ss3(argv[7]);
			T s_thres;
			ss3 >> s_thres;
			
			std::vector<T>* temp = parallel::TEEN(&data, h_top_thres, h_bot_thres, s_thres);
			data.clear();
			for (int i = 0; i<temp->size();i++){
				data.push_back(temp->at(i));
			}
		}	
	}
	else if (*argv[2] == 'd'){
		if (typeid(T) == typeid(int)){
			T pattern_arr[data.size()];
			T vals[data.size()];
	
			if (serial) {
				DSSS_encrypt(&data, pattern_arr, vals);
				DSSS_decrypt(&data, pattern_arr, vals);
			
			}
			else{
				parallel::DSSS_encrypt(&data, pattern_arr, vals);
				parallel::DSSS_decrypt(&data, pattern_arr, vals);
			}
		}
	}
	else
	{
		badsyntax = true;
	}

	// Print results
	if (*argv[3] == 'P')
	{
		printVector<T>(data);
	}
	if (*argv[3] == 'p' || *argv[3] == 'P')
	{
		if (*argv[2] == 's')
		{
			std::cout << "Sum: " << sumValue << std::endl;
		}
		else if (*argv[2] == 'a')
		{
			std::cout << "Average: " << averageValue << std::endl;				
		}
		else if (*argv[2] == 'm')
		{
			std::cout << "Min: " << minValue << std::endl;				
		}
		else if (*argv[2] == 'M')
		{
			std::cout << "Max: " << maxValue << std::endl;				
		}
		else if (*argv[2] == 'c')
		{
			std::cout << "Count: " << countValue << std::endl;
		}
		else if (*argv[2] == 'n')
		{
			std::cout << "Scan (exclusive)" << std::endl;
			printVector<T>(scanValues);
		}
		else if (*argv[2] == 'N')
		{
			std::cout << "Scan (inclusive)" << std::endl;
			printVector<T>(scanValues);
		}
		else if (*argv[2] == 't')
		{
			std::cout << "TEEN" << std::endl;
			printVector<T>(data);
		}
	}
	if (badsyntax)
	{
		printhelp();
	}
	return 0;
}

int main(int argc, char* argv[]) {
	int result = -1;
	if (argc == 2 && *argv[1] == 'h')
	{
		printhelp();
		return 0;
	}
	else if (argc >= 4)
	{
		std::vector<std::string> headerTypes;
		std::vector<std::string> headerNames;
		std::ifstream in(argv[1]);
		
		if (!in)
			return(EXIT_FAILURE);

		headerTypes = loadCSV<std::string>(in, headerTypes, -1, true);
		headerNames = loadCSV<std::string>(in, headerNames, -1, true);
		
		if (argc == 4)
		{
			if (*argv[3] == 'P')
			{
				printVector<std::string>(headerTypes);
				printVector<std::string>(headerNames);
			}
		}
		else if (argc >= 5)
		{
			int columnIndex = atoi(argv[4]);
			// std::stringstream is(*argv[4]);
			// int columnIndex;
			// is >> columnIndex;
			if (headerTypes[columnIndex].find("i") != std::string::npos)
			{
				result = mainfunc<int>(argc, argv);
			}
			else
			{
				printhelp();
			}
		}
	}
	else
	{
		printhelp();
		return (EXIT_FAILURE);
	}
	return 0;
}
