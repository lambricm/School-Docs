#include "hip/hip_runtime.h"
#include <vector>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/count.h>
#include <thrust/functional.h>
#include <thrust/sort.h>
#include <iostream>
#include <thrust/extrema.h>
#include <stdlib.h>
#include <time.h>

namespace parallel{

template <class T>
//Retrieves indicies where data from nodes should be forwarded
__global__ void TEEN_indexes(T *d_vals, int *d_ind, T* h_thres_top, T* h_thres_bot, T* s_thres){

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	
	//get indexes of values we want
	if ((d_vals[i] > *h_thres_top) || (d_vals[i] < *h_thres_bot)){
		//if the values exceed the given thresholds, collect index
		d_ind[i]= 1;
	}
	else if (i > 0){
		if (abs(d_vals[i] - d_vals[i-1]) > *s_thres){
			//or if the values changed more than the soft threshold from the last value, collect index
			d_ind[i] = 1;
		}
	}
}

template <class T>
__global__ void compact(T* d_vals, int* d_ind, T* d_vals_out){
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	if (((i == 0) && (d_ind[i] == 1)) || ((i>0) && (d_ind[i] > d_ind[i-1]))){
		d_vals_out[d_ind[i]-1] = d_vals[i];
	}
}

template <class T>
std::vector<T>* TEEN(std::vector<T>* in, T h_thres_top, T h_thres_bot, T s_thres){
	//own algorithms

	int size = in->size();
	
	T arr[size];
	std::copy(in->begin(),in->end(), arr);

	int arr_ind[size];
	for (int i = 0; i < size;i++){
		arr_ind[i] = 0;
	}

	//device values
	T *d_arr;
	T *d_h_thres_top, *d_h_thres_bot, *d_s_thres;
	int *d_arr_ind;

	//allocate space for the values & copy memory there
	hipMalloc((void**)&d_arr, size*sizeof(T));
	hipMalloc((void**)&d_arr_ind, size*sizeof(int));
	hipMalloc((void**)&d_h_thres_top, sizeof(int));
	hipMalloc((void**)&d_h_thres_bot, sizeof(int));
	hipMalloc((void**)&d_s_thres, sizeof(int));

	hipMemcpy(d_arr, arr, size * sizeof(T),hipMemcpyHostToDevice);
	hipMemcpy(d_arr_ind, arr_ind, size * sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_h_thres_top, &h_thres_top, sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_h_thres_bot, &h_thres_bot, sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_s_thres, &s_thres, sizeof(int),hipMemcpyHostToDevice);

	//get indexes for data we want
	TEEN_indexes<T><<<(size+266)/256, 256>>>(d_arr, d_arr_ind, d_h_thres_top, d_h_thres_bot, d_s_thres);

	//we can now free some values
	hipFree(d_h_thres_top);
	hipFree(d_h_thres_bot);
	hipFree(d_s_thres);

	//copy values into thrust device pointer
	thrust::device_ptr<int> d_ptr = thrust::device_malloc<int>(size);
    	thrust::copy(&d_arr_ind[0], &d_arr_ind[0]+size, d_ptr);
	thrust::device_vector<int> d_vec(d_ptr,d_ptr + size);

	//run a sum for the value count
	int sum_ind = thrust::reduce(d_vec.begin(),d_vec.end(), 0, thrust::plus<int>());
	
	//run an inclusive scan
	thrust::inclusive_scan(d_vec.begin(),d_vec.end(),d_vec.begin());
	
	//copy values back into original array
	thrust::copy(d_vec.begin(),d_vec.end(),d_arr_ind);

	//create compacted array
	T comp_array[sum_ind];
	T* d_comp_array;

	hipMalloc((void**)&d_comp_array, sum_ind*sizeof(T));
	hipMemset(&d_comp_array,  0, sum_ind*sizeof(T));

	compact<T><<<(size+266)/256, 256>>>(d_arr, d_arr_ind, d_comp_array);

	hipFree(d_arr);
	hipFree(d_arr_ind);
	
	hipMemcpy(comp_array, d_comp_array, sum_ind * sizeof(T),hipMemcpyDeviceToHost);

	hipFree(d_comp_array);

	std::vector<T> tmp(comp_array, comp_array + sum_ind);
	std::vector<T>* ret = new std::vector<T>(tmp);

	return ret;
}

template <class T>
__global__ void DSSS_operation(T* vals, T* pattern){
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	//use bitwise operator to perform xor
	vals[i] = vals[i] ^ pattern[i];
}

template <class T>
void DSSS_encrypt(std::vector<T>* in,T* pattern_arr, T* vals){
	int size = in->size();

	//get random values for encryption
	srand(time(NULL));
	for(int i = 0; i< size;i++){
		pattern_arr[i] = rand();
	}

	//copy values, prepare memory
	T *d_vals, *d_pattern;
	std::copy(in->begin(),in->end(), vals);

	hipMalloc((void**)&d_vals, size*sizeof(T));
	hipMalloc((void**)&d_pattern, size*sizeof(T));

	hipMemcpy(d_vals, vals, size*sizeof(T),hipMemcpyHostToDevice);
	hipMemcpy(d_pattern, pattern_arr, size*sizeof(T),hipMemcpyHostToDevice);

	DSSS_operation<T><<<(size+266)/256, 256>>>(d_vals, d_pattern);

	hipFree(d_pattern);

	hipMemcpy(vals, d_vals, size*sizeof(T), hipMemcpyDeviceToHost);
	hipFree(d_vals);

}

template <class T>
void DSSS_decrypt(std::vector<T>* vals, T* pattern_arr, T* in){
	int size = vals->size();

	T *d_vals, *d_pattern;

	hipMalloc((void**)&d_vals, size*sizeof(T));
	hipMalloc((void**)&d_pattern, size*sizeof(T));

	hipMemcpy(d_vals, in, size*sizeof(T),hipMemcpyHostToDevice);
	hipMemcpy(d_pattern, pattern_arr, size*sizeof(T),hipMemcpyHostToDevice);

	DSSS_operation<T><<<(size+266)/256, 256>>>(d_vals, d_pattern);

	hipFree(d_pattern);

	hipMemcpy(in, d_vals, size*sizeof(T), hipMemcpyDeviceToHost);
	hipFree(d_vals);

	std::vector<T> temp (in, in + sizeof(in) / sizeof(in[0]));
	vals = new std::vector<T>(temp);
}

template <class T>
int reduce(std::vector<T>* in){
	thrust::host_vector<T> h_vec = *in;
	thrust::device_vector<T> d_vec = h_vec;

	T sm = thrust::reduce(d_vec.begin(),d_vec.end(), 0, thrust::plus<T>());
	return sm;
}

template <class T>
float average(std::vector<T>* in){
	T sm = reduce(in);
	return sm/(static_cast<float>(in->size()));
}

template <class T>
int count(std::vector<T>* in, T &val){
	thrust::host_vector<T> h_vec = *in;
	thrust::device_vector<T> d_vec = h_vec;

	return thrust::count(d_vec.begin(),d_vec.end(), val);
}

template <class T>
void inclusive_scan(std::vector<T> &in){
	thrust::host_vector<T> h_vec = in;
	thrust::device_vector<T> d_vec = h_vec;

	thrust::inclusive_scan(d_vec.begin(),d_vec.end(),d_vec.begin());

	h_vec = d_vec;
	for (int i = 0;i < h_vec.size(); i++){
		in[i] = h_vec[i];
	}
}

template <class T>
void exclusive_scan(std::vector<T> &in){
	thrust::host_vector<T> h_vec = in;
	thrust::device_vector<T> d_vec = h_vec;

	thrust::exclusive_scan(d_vec.begin(),d_vec.end(),d_vec.begin());

	h_vec = d_vec;
	for (int i = 0;i < h_vec.size(); i++){
		in[i] = h_vec[i];
	}
}

template <class T>
int minimum(std::vector<T>* in){
	thrust::host_vector<T> h_vec = *in;
	thrust::device_vector<T> d_vec = h_vec;

	return *thrust::min_element(d_vec.begin(),d_vec.end());
}

template <class T>
int maximum(std::vector<T>* in){
	thrust::host_vector<T> h_vec = *in;
	thrust::device_vector<T> d_vec = h_vec;

	return *thrust::max_element(d_vec.begin(),d_vec.end());
}

}

template <class T>
std::vector<T>* TEEN (std::vector<T> &in, T h_thres_top, T h_thres_bot, T s_thres){
	std::vector<T>* out = new std::vector<T>();

	for (int i = 0;i < in.size();i++){

		if ((in[i] > h_thres_top) || (in[i] < h_thres_bot)){
			//if the values exceed the given thresholds, collect index
			out->push_back(in[i]);
		}
		else if (i > 0){
			if (abs(in[i] - in[i-1]) > s_thres){
				//or if the values changed more than the soft threshold from the last value, collect index
				out->push_back(in[i]);
			}
		}
	}

	return out;
}

template <class T>
void DSSS_encrypt(std::vector<T>* in, T* pattern_arr, T* vals){
	int size = in->size();

	//get random values for encryption
	srand(time(NULL));
	for(int i = 0; i< size;i++){
		pattern_arr[i] = rand();
	}

	for(int i = 0; i < size;i++){
		vals[i] = (in->at(i) ^ pattern_arr[i]);
	}
}

template <class T>
void DSSS_decrypt(std::vector<T>* in, T* pattern_arr, T* vals){
	int size = in->size();

	for(int i = 0; i < size;i++){
		vals[i] = in->at(i) ^ pattern_arr[i];	
	}
}


/* //Example Funciton calls
int main(){

	static const int arr[] = {12,13,57,91,99,123,50};
	std::vector<int> temp (arr, arr + sizeof(arr) / sizeof(arr[0]));

	std::vector<int>* vals = parallel::TEEN<int>(&temp, 100, 15, 50);

	//repetative pattern for encrypting
	int pattern_arr[temp.size()];
	int vals2[temp.size()];

	parallel::DSSS_encrypt<int>(&temp, pattern_arr, vals2);

	parallel::DSSS_decrypt<int>(&temp, pattern_arr, vals2);
	
	std::cout << parallel::maximum(&temp);
	parallel::sort(vals);

	vals = TEEN<int>(temp, 100, 15, 50);

	DSSS_encrypt<int>(&temp, pattern_arr, vals2);

	DSSS_decrypt<int>(&temp, pattern_arr, vals2);
	return 0;	
}*/
